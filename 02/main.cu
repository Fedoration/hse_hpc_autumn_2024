#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <cassert>

#define BLOCK_SIZE 16


__global__ void matrixMultiplyGlobal(const double* A, const double* B, double* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        double sum = 0.0;
        for (int k = 0; k < N; ++k) {
            sum += A[k * N + row] * B[col * N + k];
        }
        C[col * N + row] = sum;
    }
}


void matrixMultiplyCPU(const double* A, const double* B, double* C, int N) {
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            double sum = 0.0;
            for (int k = 0; k < N; ++k) {
                sum += A[k * N + row] * B[col * N + k];
            }
            C[col * N + row] = sum;
        }
    }
}


bool validateResult(const double* gpuResult, const double* cpuResult, int N, double epsilon = 1e-6) {
    for (int i = 0; i < N * N; ++i) {
        if (fabs(gpuResult[i] - cpuResult[i]) > epsilon) {
            std::cout << "Mismatch at index " << i << ": GPU result = " << gpuResult[i]
                      << ", CPU result = " << cpuResult[i] << "\n";
            return false;
        }
    }
    return true;
}


void testGlobalMemoryMatrixMultiply(int N) {
    size_t size = N * N * sizeof(double);

    double *h_A, *h_B, *h_C, *h_C_cpu;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);
    hipHostMalloc(&h_C_cpu, size, hipHostMallocDefault);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<double>(rand()) / RAND_MAX;
        h_B[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMultiplyGlobal<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Global memory multiplication time: " << milliseconds << " ms\n";
    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Проверка результата
    matrixMultiplyCPU(h_A, h_B, h_C_cpu, N);
    if (validateResult(h_C, h_C_cpu, N)) {
        std::cout << "Global memory matrix multiplication test PASSED!\n";
    } else {
        std::cout << "Global memory matrix multiplication test FAILED!\n";
    }

    // Очистка памяти
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipHostFree(h_C_cpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


void testPinnedMemoryMatrixMultiply(int N) {
    size_t size = N * N * sizeof(double);

    double *h_A, *h_B, *h_C_gpu, *h_C_cpu;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C_gpu, size, hipHostMallocDefault);
    hipHostMalloc(&h_C_cpu, size, hipHostMallocDefault);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<double>(rand()) / RAND_MAX;
        h_B[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMultiplyGlobal<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Pinned memory multiplication time: " << milliseconds << " ms\n";

    hipMemcpy(h_C_gpu, d_C, size, hipMemcpyDeviceToHost);

    // Проверка результата
    matrixMultiplyCPU(h_A, h_B, h_C_cpu, N);
    if (validateResult(h_C_gpu, h_C_cpu, N)) {
        std::cout << "Pinned memory test PASSED!\n";
    } else {
        std::cout << "Pinned memory test FAILED!\n";
    }

    // Очистка памяти
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C_gpu);
    hipHostFree(h_C_cpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


void testUnifiedMemoryMatrixMultiply(int N) {
    size_t size = N * N * sizeof(double);

    double *A, *B, *C;
    hipMallocManaged(&A, size);
    hipMallocManaged(&B, size);
    hipMallocManaged(&C, size);

    for (int i = 0; i < N * N; ++i) {
        A[i] = static_cast<double>(rand()) / RAND_MAX;
        B[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMultiplyGlobal<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Unified memory multiplication time: " << milliseconds << " ms\n";

    hipDeviceSynchronize();

    // Проверка результата
    double *C_cpu = new double[N * N];
    matrixMultiplyCPU(A, B, C_cpu, N);
    if (validateResult(C, C_cpu, N)) {
        std::cout << "Unified memory test PASSED!\n";
    } else {
        std::cout << "Unified memory test FAILED!\n";
    }

    // Очистка памяти
    hipFree(A);
    hipFree(B);
    hipFree(C);
    delete[] C_cpu;
}


void testStreamsMatrixMultiply(int N, int numStreams) {
    size_t size = N * N * sizeof(double);
    size_t chunkSize = (N / numStreams) * N;

    double *h_A, *h_B, *h_C, *h_C_cpu;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);
    hipHostMalloc(&h_C_cpu, size, hipHostMallocDefault);

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<double>(rand()) / RAND_MAX;
        h_B[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    for (int i = 0; i < numStreams; ++i) {
        size_t offset = i * chunkSize;
        hipMemcpyAsync(d_A + offset, h_A + offset, chunkSize * sizeof(double), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_B + offset, h_B + offset, chunkSize * sizeof(double), hipMemcpyHostToDevice, streams[i]);
    }

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    matrixMultiplyGlobal<<<blocksPerGrid, threadsPerBlock, 0, streams[0]>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Streams multiplication time: " << milliseconds << " ms\n";

    for (int i = 0; i < numStreams; ++i) {
        size_t offset = i * chunkSize;
        hipMemcpyAsync(h_C + offset, d_C + offset, chunkSize * sizeof(double), hipMemcpyDeviceToHost, streams[i]);
    }

    for (int i = 0; i < numStreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    for (int i = 0; i < numStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    // Проверка результата
    matrixMultiplyCPU(h_A, h_B, h_C_cpu, N);
    if (validateResult(h_C, h_C_cpu, N)) {
        std::cout << "Streams test PASSED!\n";
    } else {
        std::cout << "Streams test FAILED!\n";
    }

    // Очистка памяти
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipHostFree(h_C_cpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


__global__ void matrixMulShared(double* A, double* B, double* C, int N) {
    __shared__ double Asub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double Bsub[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * BLOCK_SIZE + ty;
    int col = blockIdx.x * BLOCK_SIZE + tx;

    double Cvalue = 0.0f;

    for (int t = 0; t < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; t++) {
        if (row < N && t * BLOCK_SIZE + tx < N)
            Asub[ty][tx] = A[(t * BLOCK_SIZE + tx) * N + row];
        else
            Asub[ty][tx] = 0.0f;

        if (col < N && t * BLOCK_SIZE + ty < N)
            Bsub[ty][tx] = B[col * N + t * BLOCK_SIZE + ty]; 
        else
            Bsub[ty][tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; k++) {
            Cvalue += Asub[ty][k] * Bsub[k][tx];
        }

        __syncthreads();
    }

    if (row < N && col < N)
        C[col * N + row] = Cvalue;
}


void testSharedMemoryMatrixMultiply(int N) {
    size_t size = N * N * sizeof(double);

    double *h_A, *h_B, *h_C, *h_C_cpu;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);
    hipHostMalloc(&h_C_cpu, size, hipHostMallocDefault);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<double>(rand()) / RAND_MAX;
        h_B[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    matrixMulShared<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Shared memory multiplication time: " << milliseconds << " ms\n";

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Проверка результата
    matrixMultiplyCPU(h_A, h_B, h_C_cpu, N);
    if (validateResult(h_C, h_C_cpu, N)) {
        std::cout << "Shared memory test PASSED!\n";
    } else {
        std::cout << "Shared memory test FAILED!\n";
    }

    // Очистка памяти
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipHostFree(h_C_cpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


void checkCudaStatus(hipError_t status, const char* msg) {
    if (status != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublasStatus(hipblasStatus_t status, const char* msg) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS Error: " << msg << std::endl;
        exit(EXIT_FAILURE);
    }
}


void testCublasMatrixMultiply(int N) {
    size_t size = N * N * sizeof(float);

    float *h_A, *h_B, *h_C, *h_C_cpu;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);
    hipHostMalloc(&h_C_cpu, size, hipHostMallocDefault);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    checkCublasStatus(hipblasCreate(&handle), "Failed to create cuBLAS handle");

    const float alpha = 1.0f; 
    const float beta = 0.0f;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Выполняем умножение матриц: C = alpha * A * B + beta * C
    checkCublasStatus(
        hipblasSgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,  
                    N, N, N,                   
                    &alpha,                    
                    d_A, N,                    
                    d_B, N,                    
                    &beta,                     
                    d_C, N),                   
        "Failed to execute hipblasSgemm"
    );

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Cublas multiplication time: " << milliseconds << " ms\n"; 

    checkCudaStatus(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost), "Failed to copy C to host");

    checkCublasStatus(hipblasDestroy(handle), "Failed to destroy cuBLAS handle");
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


int main() {
    int N = 1024;

    testGlobalMemoryMatrixMultiply(N);
    testPinnedMemoryMatrixMultiply(N);
    testUnifiedMemoryMatrixMultiply(N);
    testStreamsMatrixMultiply(N, 4);
    testSharedMemoryMatrixMultiply(N);
    testCublasMatrixMultiply(N);
}
